#include "hip/hip_runtime.h"
__global__ void addVectors(float *result, float *a, float *b, int N){
    int index = threadIdx.x + blockIdx.x + *blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < N; i += stride){
        result[i] = a[i] + b[i];
    }
}