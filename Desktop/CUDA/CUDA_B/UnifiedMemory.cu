//통합 메모리(Unified Memory) 할당
float *a;
float *b;
float *c;

hipMallocManaged(&a, size);
hipMallocManaged(&b, size);
hipMallocManaged(&c, size);

hipMemPrefetchAsync(a, size, deviceId);
hipMemPrefetchAsync(b, size, deviceId);
hipMemPrefetchAsync(c, size, deviceId);

//커널을 이용한 초기화
threadsPerBlock = 256;
numberOfBlocks = 32 * numberOfSMs;
initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);
addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
//에러 확인, GPU 기다림, 실행여부 점검
addVectorsErr = hipGetLastError();
if(addVectorsErr != hipSuccess) printf("Error: %s\n",hipGetErrorString(addVectorsErr));

asyncErr = hipDeviceSynchronize();
if(asyncErr != hipSuccess) printf("Error: %s\n",hipGetErrorString(asyncErr));

checkElementsAre(7, c, N);
hipFree(a);
hipFree(b);
hipFree(c);
