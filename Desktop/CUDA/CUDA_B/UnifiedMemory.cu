float *a;
float *b;
float *c;

hipMallocManaged(&a, size);
hipMallocManaged(&b, size);
hipMallocManaged(&c, size);

hipMemPrefetchAsync(a, size, deviceId);
hipMemPrefetchAsync(b, size, deviceId);
hipMemPrefetchAsync(c, size, deviceId);